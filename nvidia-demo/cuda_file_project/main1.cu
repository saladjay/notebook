// main.cu
#include <iostream>
#include <hip/hip_runtime.h>
// #include "kernel1.cuh"

extern void __global__ kernel(float* data, int size);

int main() {
    const int arraySize = 5;
    float data[arraySize] = {1.0f, 2.0f, 5.0f, 4.0f, 5.0f};

    float* dev_data;
    hipMalloc((void**)&dev_data, arraySize * sizeof(float));
    hipMemcpy(dev_data, data, arraySize * sizeof(float), hipMemcpyHostToDevice);

    kernel<<<1, arraySize>>>(dev_data, arraySize);
    hipMemcpy(data, dev_data, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_data);

    for (int i = 0; i < arraySize; ++i) {
        std::cout << data[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}